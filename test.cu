#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void hello()
{
	printf("Hello world\n");
}


int main() {
	//printf("Hello world\n");
	//int* a;
	//hipMalloc(&a, 100);
	//hipFree(a);

	hello << <1,1023 >> > ();
	hipDeviceSynchronize();

	return 0;
}